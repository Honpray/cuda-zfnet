#include <stdio.h>
#include <stdlib.h>

#include "zf_kernel.cu"

const int INPUT_SIZE = 224 * 224 * 3;

const int LAYER_1_INPUT_SIZE = 110 * 110 * 96;
const int LAYER_1_FILTER_SIZE = 7 * 7 * 3;
const int LAYER_1_FILTER_NUM = 96;
const int LAYER_1_POOLED_SIZE = 55 * 55 * 96;
const int LAYER_1_PADDED_SIZE = 57 * 57 * 96;

const int LAYER_2_INPUT_SIZE = 26 * 26 * 256;
const int LAYER_2_FILTER_SIZE = 5 * 5 * 96;
const int LAYER_2_FILTER_NUM = 256;
const int LAYER_2_POOLED_SIZE = 13 * 13 * 256;
const int LAYER_2_PADDED_SIZE = 15 * 15 * 256;

const int LAYER_3_INPUT_SIZE = 13 * 13 * 384;
const int LAYER_3_FILTER_SIZE = 3 * 3 * 256;
const int LAYER_3_FILTER_NUM = 384;

const int LAYER_4_INPUT_SIZE = 13 * 13 * 384;
const int LAYER_4_FILTER_SIZE = 3 * 3 * 384;
const int LAYER_4_FILTER_NUM = 384;

const int LAYER_5_INPUT_SIZE = 13 * 13 * 256;
const int LAYER_5_POOLED_SIZE = 6 * 6 * 256;

void read_file(const char *file_path, float *dest_array);

int main(int argc, char **argv) {

  // host
  float *input_array; // layer_1_input
  float *layer_1_weights;
  float *layer_2_weights;

  input_array = (float *)malloc(INPUT_SIZE * sizeof(float));
  layer_1_weights = (float *)malloc(LAYER_1_FILTER_SIZE * LAYER_1_FILTER_NUM * sizeof(float));
  layer_2_weights = (float *)malloc(LAYER_2_FILTER_SIZE * LAYER_2_FILTER_NUM * sizeof(float));

  // read_file("data/input.txt", input_array);
  read_file("data/layer1.txt", layer_1_weights);
  read_file("data/layer2.txt", layer_2_weights);



  // device 
  float *d_input;
  float *d_layer_1_input;
  float *d_layer_1_weights;
  float *d_layer_1_pooled;
  float *d_layer_1_padded;

  hipMalloc((void **)&d_input, INPUT_SIZE * sizeof(float));
  hipMalloc((void **)&d_layer_1_input, LAYER_1_INPUT_SIZE * sizeof(float));

  hipMalloc((void **)&d_layer_1_weights, LAYER_1_FILTER_SIZE * LAYER_1_FILTER_NUM * sizeof(float));
  hipMemcpy(d_layer_1_weights, layer_1_weights, LAYER_1_FILTER_SIZE * LAYER_1_FILTER_NUM * sizeof(float), hipMemcpyHostToDevice);

  hipMalloc((void **)&d_layer_1_pooled, LAYER_1_POOLED_SIZE * sizeof(float));
  
  hipMalloc((void **)&d_layer_1_padded, LAYER_1_PADDED_SIZE * sizeof(float));
  hipMemset(d_layer_1_padded, 0, LAYER_1_POOLED_SIZE * sizeof(float));



  // layer 1: 110 * 110 * 96
  dim3 conv_1_grid_dim(96, 1, 1);
  dim3 conv_1_block_dim(110, 110);

  printf("Running conv_1 ...\n");
  run_conv_1<<<conv_1_grid_dim, conv_1_block_dim>>>(d_input, d_layer_1_weights, d_layer_1_input);

  dim3 pool_1_grid_dim(96, 1, 1);
  dim3 pool_1_block_dim(55, 55);
  printf("Running pool_1 ...\n");
  run_pool_1<<<pool_1_grid_dim, pool_1_block_dim>>>(d_layer_1_input, d_layer_1_pooled);

  dim3 pad_1_grid_dim(96, 1, 1);
  dim3 pad_1_block_dim(55, 55);
  printf("Padding pool_1 output ...\n");
  run_padding_1<<<pad_1_grid_dim, pad_1_block_dim>>>(d_layer_1_pooled, d_layer_1_padded);

  dim3 lcn_1_grid_dim(96, 1, 1);
  dim3 lcn_1_block_dim(55, 55);
  printf("Running lcn_1\n");
  run_lcn_1<<<lcn_1_grid_dim, lcn_1_block_dim>>>(d_layer_1_padded);
}


void read_file(const char *file_path, float *dest_array) {
  FILE *fp = fopen(file_path, "r");
  int count = 0;
  char *line = NULL;
  size_t len = 0;
  ssize_t nread;
  if (fp == NULL) {
    perror("fopen");
    exit(EXIT_FAILURE);
  }
  while ((nread = getline(&line, &len, fp)) != -1) {
    printf("Read %zu\n", len);
    dest_array[count++] = atof(line);
  }
  free(line);
  fclose(fp);
}