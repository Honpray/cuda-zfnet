#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "zf_kernel.cu"

const int INPUT_SIZE = 224 * 224 * 3;

const int LAYER_1_INPUT_SIZE = 110 * 110 * 96;
const int LAYER_1_FILTER_SIZE = 7 * 7 * 3;
const int LAYER_1_FILTER_NUM = 96;
const int LAYER_1_POOLED_SIZE = 55 * 55 * 96;
const int LAYER_1_PADDED_SIZE = 57 * 57 * 96;

const int LAYER_2_INPUT_SIZE = 26 * 26 * 256;
const int LAYER_2_FILTER_SIZE = 5 * 5 * 96;
const int LAYER_2_FILTER_NUM = 256;
const int LAYER_2_POOLED_SIZE = 13 * 13 * 256;
const int LAYER_2_PADDED_SIZE = 15 * 15 * 256;

const int LAYER_3_INPUT_SIZE = 13 * 13 * 384;
const int LAYER_3_FILTER_SIZE = 3 * 3 * 256;
const int LAYER_3_FILTER_NUM = 384;

const int LAYER_4_INPUT_SIZE = 13 * 13 * 384;
const int LAYER_4_FILTER_SIZE = 3 * 3 * 384;
const int LAYER_4_FILTER_NUM = 384;

const int LAYER_5_INPUT_SIZE = 13 * 13 * 256;
const int LAYER_5_POOLED_SIZE = 6 * 6 * 256;
const int LAYER_5_FILTER_SIZE = 3 * 3 * 384;
const int LAYER_5_FILTER_NUM = 256;

const int LAYER_6_INPUT_SIZE = 1 * 1 * 4096;
const int LAYER_6_FILTER_SIZE = 6 * 6 * 256;
const int LAYER_6_FILTER_NUM = 4096;

const int LAYER_7_INPUT_SIZE = 1 * 1 * 4096;
const int LAYER_7_FILTER_SIZE = 1 * 1 * 4096;
const int LAYER_7_FILTER_NUM = 4096;

const int LAYER_8_INPUT_SIZE = 1 * 1 * 1000;
const int LAYER_8_FILTER_SIZE = 1 * 1 * 4096;
const int LAYER_8_FILTER_NUM = 1000;

const int OUTPUT_SIZE = 1000;

void read_file(const char *file_path, float *dest_array);

int main(int argc, char **argv) {

  // host
  float *input_array; // layer_1_input
  float *layer_1_weights;
  float *layer_2_weights;
  float *layer_3_weights;
  float *layer_4_weights;
  float *layer_5_weights;
  float *layer_6_weights;
  float *layer_7_weights;
  float *layer_8_weights;
  float *output_array;

  input_array = (float *)malloc(INPUT_SIZE * sizeof(float));

  layer_1_weights = (float *)malloc(LAYER_1_FILTER_SIZE * LAYER_1_FILTER_NUM * sizeof(float));
  layer_2_weights = (float *)malloc(LAYER_2_FILTER_SIZE * LAYER_2_FILTER_NUM * sizeof(float));
  layer_3_weights = (float *)malloc(LAYER_3_FILTER_SIZE * LAYER_3_FILTER_NUM * sizeof(float));
  layer_4_weights = (float *)malloc(LAYER_4_FILTER_SIZE * LAYER_4_FILTER_NUM * sizeof(float));
  layer_5_weights = (float *)malloc(LAYER_5_FILTER_SIZE * LAYER_5_FILTER_NUM * sizeof(float));
  layer_6_weights = (float *)malloc(LAYER_6_FILTER_SIZE * LAYER_6_FILTER_NUM * sizeof(float));
  layer_7_weights = (float *)malloc(LAYER_7_FILTER_SIZE * LAYER_7_FILTER_NUM * sizeof(float));
  layer_8_weights = (float *)malloc(LAYER_8_FILTER_SIZE * LAYER_8_FILTER_NUM * sizeof(float));

  output_array = (float *)malloc(OUTPUT_SIZE * sizeof(float));

  read_file("data/input.txt", input_array);


  printf("Reading layer1 weights ...\n");
  read_file("data/layer1.txt", layer_1_weights);
  printf("Reading layer2 weights ...\n");
  read_file("data/layer2.txt", layer_2_weights);
  printf("Reading layer3 weights ...\n");
  read_file("data/layer3.txt", layer_3_weights);
  printf("Reading layer4 weights ...\n");
  read_file("data/layer4.txt", layer_4_weights);
  printf("Reading layer5 weights ...\n");
  read_file("data/layer5.txt", layer_5_weights);
  printf("Reading layer6 weights ...\n");
  read_file("data/layer6.txt", layer_6_weights);
  printf("Reading layer7 weights ...\n");
  read_file("data/layer7.txt", layer_7_weights);
  printf("Reading layer8 weights ...\n");
  read_file("data/layer8.txt", layer_8_weights);



  // device
  float *d_input;
  float *d_layer_1_input;
  float *d_layer_1_weights;
  float *d_layer_1_pooled;
  float *d_layer_1_padded;

  float *d_layer_2_input;
  float *d_layer_2_weights;
  float *d_layer_2_pooled;
  float *d_layer_2_padded;

  float *d_layer_3_input;
  float *d_layer_3_weights;

  float *d_layer_4_input;
  float *d_layer_4_weights;

  float *d_layer_5_input;
  float *d_layer_5_pooled;
  float *d_layer_5_weights;

  float *d_layer_6_input;
  float *d_layer_6_weights;

  float *d_layer_7_input;
  float *d_layer_7_weights;

  float *d_layer_8_input;
  float *d_layer_8_weights;

  hipError_t err_code;


  // input
  hipMalloc((void **)&d_input, INPUT_SIZE * sizeof(float));
  hipMemcpy(d_input, input_array, INPUT_SIZE* sizeof(float), hipMemcpyHostToDevice);

  // layer 1
  hipMalloc((void **)&d_layer_1_input, LAYER_1_INPUT_SIZE * sizeof(float));
  hipMalloc((void **)&d_layer_1_weights, LAYER_1_FILTER_SIZE * LAYER_1_FILTER_NUM * sizeof(float));
  hipMemcpy(d_layer_1_weights, layer_1_weights, LAYER_1_FILTER_SIZE * LAYER_1_FILTER_NUM * sizeof(float), hipMemcpyHostToDevice);

  hipMalloc((void **)&d_layer_1_pooled, LAYER_1_POOLED_SIZE * sizeof(float));

  hipMalloc((void **)&d_layer_1_padded, LAYER_1_PADDED_SIZE * sizeof(float));
  hipMemset(d_layer_1_padded, 0, LAYER_1_POOLED_SIZE * sizeof(float));

  // layer 2
  hipMalloc((void **)&d_layer_2_input, LAYER_2_INPUT_SIZE * sizeof(float));

  hipMalloc((void **)&d_layer_2_weights, LAYER_2_FILTER_SIZE * LAYER_2_FILTER_NUM * sizeof(float));
  hipMemcpy(d_layer_2_weights, layer_2_weights, LAYER_2_FILTER_SIZE * LAYER_2_FILTER_NUM * sizeof(float), hipMemcpyHostToDevice);

  hipMalloc((void **)&d_layer_2_pooled, LAYER_2_POOLED_SIZE * sizeof(float));

  hipMalloc((void **)&d_layer_2_padded, LAYER_2_PADDED_SIZE * sizeof(float));
  hipMemset(d_layer_2_padded, 0, LAYER_2_POOLED_SIZE * sizeof(float));


  // layer 3
  hipMalloc((void **)&d_layer_3_input, LAYER_3_INPUT_SIZE * sizeof(float));

  hipMalloc((void **)&d_layer_3_weights, LAYER_3_FILTER_SIZE * LAYER_3_FILTER_NUM * sizeof(float));
  hipMemcpy(d_layer_3_weights, layer_3_weights, LAYER_3_FILTER_SIZE * LAYER_3_FILTER_NUM * sizeof(float), hipMemcpyHostToDevice);

  // layer 4
  hipMalloc((void **)&d_layer_4_input, LAYER_4_INPUT_SIZE * sizeof(float));

  hipMalloc((void **)&d_layer_4_weights, LAYER_4_FILTER_SIZE * LAYER_4_FILTER_NUM * sizeof(float));
  hipMemcpy(d_layer_4_weights, layer_4_weights, LAYER_4_FILTER_SIZE * LAYER_4_FILTER_NUM * sizeof(float), hipMemcpyHostToDevice);


  // layer 5
  hipMalloc((void **)&d_layer_5_input, LAYER_5_INPUT_SIZE * sizeof(float));

  hipMalloc((void **)&d_layer_5_weights, LAYER_5_FILTER_SIZE * LAYER_5_FILTER_NUM * sizeof(float));
  hipMemcpy(d_layer_5_weights, layer_5_weights, LAYER_5_FILTER_SIZE * LAYER_5_FILTER_NUM * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **)&d_layer_5_pooled, LAYER_5_POOLED_SIZE * sizeof(float));


  // layer 6
  hipMalloc((void **)&d_layer_6_input, LAYER_6_INPUT_SIZE * sizeof(float));

  hipMalloc((void **)&d_layer_6_weights, LAYER_6_FILTER_SIZE * LAYER_6_FILTER_NUM * sizeof(float));
  hipMemcpy(d_layer_6_weights, layer_6_weights, LAYER_4_FILTER_SIZE * LAYER_4_FILTER_NUM * sizeof(float), hipMemcpyHostToDevice);


  // layer 7
  hipMalloc((void **)&d_layer_7_input, LAYER_7_INPUT_SIZE * sizeof(float));

  hipMalloc((void **)&d_layer_7_weights, LAYER_7_FILTER_SIZE * LAYER_7_FILTER_NUM * sizeof(float));
  hipMemcpy(d_layer_7_weights, layer_7_weights, LAYER_7_FILTER_SIZE * LAYER_7_FILTER_NUM * sizeof(float), hipMemcpyHostToDevice);


  // layer 8
  hipMalloc((void **)&d_layer_8_input, LAYER_8_INPUT_SIZE * sizeof(float));

  hipMalloc((void **)&d_layer_8_weights, LAYER_8_FILTER_SIZE * LAYER_8_FILTER_NUM * sizeof(float));
  hipMemcpy(d_layer_8_weights, layer_8_weights, LAYER_8_FILTER_SIZE * LAYER_8_FILTER_NUM * sizeof(float), hipMemcpyHostToDevice);




  // layer 1: 110 * 110 * 96
  printf("Running conv_1 ...\n");
  dim3 conv_1_grid_dim00(96, 1, 1);
  dim3 conv_1_block_dim00(32, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim00, conv_1_block_dim00>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 0, 0);


  dim3 conv_1_grid_dim01(96, 1, 1);
  dim3 conv_1_block_dim01(32, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim01, conv_1_block_dim01>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 0, 32);

  dim3 conv_1_grid_dim02(96, 1, 1);
  dim3 conv_1_block_dim02(32, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim02, conv_1_block_dim02>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 0, 64);

  dim3 conv_1_grid_dim03(96, 1, 1);
  dim3 conv_1_block_dim03(32, 14);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim03, conv_1_block_dim03>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 0, 96);

  dim3 conv_1_grid_dim10(96, 1, 1);
  dim3 conv_1_block_dim10(32, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim10, conv_1_block_dim10>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 32, 0);

  dim3 conv_1_grid_dim11(96, 1, 1);
  dim3 conv_1_block_dim11(32, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim11, conv_1_block_dim11>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 32, 32);

  dim3 conv_1_grid_dim12(96, 1, 1);
  dim3 conv_1_block_dim12(32, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim12, conv_1_block_dim12>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 32, 64);

  dim3 conv_1_grid_dim13(96, 1, 1);
  dim3 conv_1_block_dim13(32, 14);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim13, conv_1_block_dim13>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 32, 96);

  dim3 conv_1_grid_dim20(96, 1, 1);
  dim3 conv_1_block_dim20(32, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim20, conv_1_block_dim20>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 64, 0);

  dim3 conv_1_grid_dim21(96, 1, 1);
  dim3 conv_1_block_dim21(32, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim21, conv_1_block_dim21>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 64, 32);

  dim3 conv_1_grid_dim22(96, 1, 1);
  dim3 conv_1_block_dim22(32, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim22, conv_1_block_dim22>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 64, 64);

  dim3 conv_1_grid_dim23(96, 1, 1);
  dim3 conv_1_block_dim23(32, 14);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim23, conv_1_block_dim23>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 64, 96);

  dim3 conv_1_grid_dim30(96, 1, 1);
  dim3 conv_1_block_dim30(14, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim30, conv_1_block_dim30>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 96, 0);

  dim3 conv_1_grid_dim31(96, 1, 1);
  dim3 conv_1_block_dim31(14, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim31, conv_1_block_dim31>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 96, 32);

  dim3 conv_1_grid_dim32(96, 1, 1);
  dim3 conv_1_block_dim32(14, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim32, conv_1_block_dim32>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 96, 64);

  dim3 conv_1_grid_dim33(96, 1, 1);
  dim3 conv_1_block_dim33(14, 32);
  // stride 2, filter size 7, channel_num 3, input_width 224, output_width 110
  run_conv<<<conv_1_grid_dim33, conv_1_block_dim33>>>(d_input, d_layer_1_weights, d_layer_1_input, 2, 7, 3, 224, 110, 96, 96);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("conv error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }

  printf("Running pool_1 ...\n");
  dim3 pool_1_grid_dim00(96, 1, 1);
  dim3 pool_1_block_dim00(32, 32);
  // stride 2, pool size 3, input_width 110, output_width 55
  run_pool<<<pool_1_grid_dim00, pool_1_block_dim00>>>(d_layer_1_input, d_layer_1_pooled, 2, 3, 110, 55, 0, 0);

  dim3 pool_1_grid_dim01(96, 1, 1);
  dim3 pool_1_block_dim01(32, 23);
  // stride 2, pool size 3, input_width 110, output_width 55
  run_pool<<<pool_1_grid_dim01, pool_1_block_dim01>>>(d_layer_1_input, d_layer_1_pooled, 2, 3, 110, 55, 0, 32);

  dim3 pool_1_grid_dim10(96, 1, 1);
  dim3 pool_1_block_dim10(23, 32);
  // stride 2, pool size 3, input_width 110, output_width 55
  run_pool<<<pool_1_grid_dim10, pool_1_block_dim10>>>(d_layer_1_input, d_layer_1_pooled, 2, 3, 110, 55, 32, 0);

  dim3 pool_1_grid_dim11(96, 1, 1);
  dim3 pool_1_block_dim11(23, 23);
  // stride 2, pool size 3, input_width 110, output_width 55
  run_pool<<<pool_1_grid_dim11, pool_1_block_dim11>>>(d_layer_1_input, d_layer_1_pooled, 2, 3, 110, 55, 32, 32);

  printf("Padding pool_1 output ...\n");
  dim3 pad_1_grid_dim00(96, 1, 1);
  dim3 pad_1_block_dim00(32, 32);
  // width 55
  run_padding<<<pad_1_grid_dim00, pad_1_block_dim00>>>(d_layer_1_pooled, d_layer_1_padded, 55, 0, 0);

  dim3 pad_1_grid_dim01(96, 1, 1);
  dim3 pad_1_block_dim01(32, 23);
  // width 55
  run_padding<<<pad_1_grid_dim01, pad_1_block_dim01>>>(d_layer_1_pooled, d_layer_1_padded, 55, 0 ,32);

  dim3 pad_1_grid_dim10(96, 1, 1);
  dim3 pad_1_block_dim10(23, 32);
  // width 55
  run_padding<<<pad_1_grid_dim10, pad_1_block_dim10>>>(d_layer_1_pooled, d_layer_1_padded, 55, 32, 0);

  dim3 pad_1_grid_dim11(96, 1, 1);
  dim3 pad_1_block_dim11(23, 23);
  // width 55
  run_padding<<<pad_1_grid_dim11, pad_1_block_dim11>>>(d_layer_1_pooled, d_layer_1_padded, 55, 32, 32);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("pooling error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }

  printf("Running lcn_1 ...\n");
  dim3 lcn_1_grid_dim00(96, 1, 1);
  dim3 lcn_1_block_dim00(32, 32);
  // width 55
  run_lcn<<<lcn_1_grid_dim00, lcn_1_block_dim00>>>(d_layer_1_padded, d_layer_1_pooled, 55, 0, 0);

  dim3 lcn_1_grid_dim01(96, 1, 1);
  dim3 lcn_1_block_dim01(32, 23);
  // width 55
  run_lcn<<<lcn_1_grid_dim01, lcn_1_block_dim01>>>(d_layer_1_padded, d_layer_1_pooled, 55, 0, 32);

  dim3 lcn_1_grid_dim10(96, 1, 1);
  dim3 lcn_1_block_dim10(23, 32);
  // width 55
  run_lcn<<<lcn_1_grid_dim10, lcn_1_block_dim10>>>(d_layer_1_padded, d_layer_1_pooled, 55, 32, 0);

  dim3 lcn_1_grid_dim11(96, 1, 1);
  dim3 lcn_1_block_dim11(23, 23);
  // width 55
  run_lcn<<<lcn_1_grid_dim11, lcn_1_block_dim11>>>(d_layer_1_padded, d_layer_1_pooled, 55, 32, 32);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("lcn error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }


  // layer 2: 26 * 26 * 256
  dim3 conv_2_grid_dim(256, 1, 1);
  dim3 conv_2_block_dim(26, 26);

  printf("Running conv_2 ...\n");
  // stride 2, filter size 5, channel_num 96, input_width 55, output_width 26
  run_conv<<<conv_2_grid_dim, conv_2_block_dim>>>(d_layer_1_pooled, d_layer_2_weights, d_layer_2_input, 2, 5, 96, 55, 26, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("conv2 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }

  dim3 pool_2_grid_dim(256, 1, 1);
  dim3 pool_2_block_dim(26, 26);
  printf("Running pool_2 ...\n");
  // stride 2, pool size 3, input_width 26, output_width 13
  run_pool<<<pool_2_grid_dim, pool_2_block_dim>>>(d_layer_2_input, d_layer_2_pooled, 2, 3, 26, 13, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("pool2 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }

  dim3 pad_2_grid_dim(256, 1, 1);
  dim3 pad_2_block_dim(26, 26);
  printf("Padding pool_2 output ...\n");
  // width 26
  run_padding<<<pad_2_grid_dim, pad_2_block_dim>>>(d_layer_2_pooled, d_layer_2_padded, 26, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("padding2 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }

  dim3 lcn_2_grid_dim(256, 1, 1);
  dim3 lcn_2_block_dim(26, 26);
  printf("Running lcn_2 ...\n");
  // width 13
  run_lcn<<<lcn_2_grid_dim, lcn_2_block_dim>>>(d_layer_2_padded, d_layer_2_pooled, 13, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("lcn2 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }


  // layer 3: 13 * 13 * 384
  dim3 conv_3_grid_dim(384, 1, 1);
  dim3 conv_3_block_dim(13, 13);

  printf("Running conv_3 ...\n");
  // stride 1, filter size 3, channel_num 256, input_width 13,  output_width 13
  run_conv<<<conv_3_grid_dim, conv_3_block_dim>>>(d_layer_2_pooled, d_layer_3_weights, d_layer_3_input, 1, 3, 256, 13, 13, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("conv3 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }


  // layer 4: 13 * 13 * 384
  dim3 conv_4_grid_dim(384, 1, 1);
  dim3 conv_4_block_dim(13, 13);

  printf("Running conv_4 ...\n");
  // stride 1, filter size 3, channel_num 384, input_width 13,  output_width 13
  run_conv<<<conv_4_grid_dim, conv_4_block_dim>>>(d_layer_3_input, d_layer_4_weights, d_layer_4_input, 1, 3, 384, 13, 13, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("conv4 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }

  // layer 5: 13 * 13 * 256
  dim3 conv_5_grid_dim(256, 1, 1);
  dim3 conv_5_block_dim(13, 13);

  printf("Running conv_5 ...\n");
  // stride 1, filter size 3, channel_num 384, input_width 13,  output_width 13
  run_conv<<<conv_5_grid_dim, conv_5_block_dim>>>(d_layer_4_input, d_layer_5_weights, d_layer_5_input, 1, 3, 384, 13, 13, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("conv5 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }

  dim3 pool_5_grid_dim(256, 1, 1);
  dim3 pool_5_block_dim(13, 13);
  printf("Running pool_5 ...\n");
  // stride 2, pool size 3, input_width 13, output_width 6
  run_pool<<<pool_5_grid_dim, pool_5_block_dim>>>(d_layer_5_input, d_layer_5_pooled, 2, 3, 13, 6, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("pool5 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }


  // layer 6: 1 * 1 * 4096
  dim3 fc_6_grid_dim(4096, 1, 1);
  dim3 fc_6_block_dim(1, 1);

  printf("Running fc_6 ...\n");
  // stride 1, filter size 1, channel_num 256, input_width 6,  output_width 1
  run_conv<<<fc_6_grid_dim, fc_6_block_dim>>>(d_layer_5_pooled, d_layer_6_weights, d_layer_6_input, 1, 1, 256, 6, 1, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("fc6 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }


  // layer 7: 1 * 1 * 4096
  dim3 fc_7_grid_dim(4096, 1, 1);
  dim3 fc_7_block_dim(1, 1);

  printf("Running fc_7 ...\n");
  // stride 1, filter size 1, channel_num 4096, input_width 1,  output_width 1
  run_conv<<<fc_7_grid_dim, fc_7_block_dim>>>(d_layer_6_input, d_layer_7_weights, d_layer_7_input, 1, 1, 4096, 1, 1, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("fc7 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }


  // layer 8: 1 * 1 * 1000
  dim3 fc_8_grid_dim(1000, 1, 1);
  dim3 fc_8_block_dim(1, 1);

  printf("Running fc_8 ...\n");
  // stride 1, filter size 1, channel_num 4096, input_width 1,  output_width 1
  run_fc_8<<<fc_8_grid_dim, fc_8_block_dim>>>(d_layer_7_input, d_layer_8_weights, d_layer_8_input, 1, 1, 4096, 1, 1, 0, 0);

  err_code = hipGetLastError();
  if (err_code != hipSuccess) {
    printf("fc8 error: %s\n", hipGetErrorString(err_code));
    exit(EXIT_FAILURE);
  }

  // extra relu

  hipMemcpy(output_array, d_layer_8_input, OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);

  float max = 0;
  int output_index;
  for (int i = 0; i < 1000; i++) {
    // printf("%f\n", (output_array[i]));
    if (max < output_array[i]) {
      max = output_array[i];
      output_index = i;
    }
  }
  printf("Index: %d\n", output_index);
}


void read_file(const char *file_path, float *dest_array) {
  FILE *fp = fopen(file_path, "r");
  int count = 0;
  char *line = NULL;
  size_t len = 0;
  ssize_t nread;
  if (fp == NULL) {
    perror("fopen");
    exit(EXIT_FAILURE);
  }
  while ((nread = getline(&line, &len, fp)) != -1) {
    // printf("Read %zu\n", len);
    dest_array[count++] = atof(line);
  }
  free(line);
  fclose(fp);
}
