
#include <hip/hip_runtime.h>
__global__ void run_conv_1(float *d_layer_1_input, float *d_layer_1_weights, float *d_layer_2_input) {
  float product = 0;
  int stride = 2;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  for (int i = 0; i < 7; i++) {
    for (int j = 0; j < 7; j++) {
      product += d_layer_1_input[3 * (i * 224 + j + tx * stride + ty * 224 * stride)] * d_layer_1_weights[i * 11 + j + bx * 11 * 11 * 3 + 11 * 11];
      product += d_layer_1_input[3 * (i * 224 + j + tx * stride + ty * 224 * stride) + 1] * d_layer_1_weights[i * 11 + j + bx * 11 * 11 * 3 + 11 * 11 * 2];
      product += d_layer_1_input[3 * (i * 224 + j + tx * stride + ty * 224 * stride) + 2] * d_layer_1_weights[i * 11 + j + bx * 11 * 11 * 3 + 11 * 11 * 3];
    }
  }
  // ReLU
  if (product < 0) {
    product = 0;
  }
  d_layer_2_input[tx + ty * 110 + bx * 110 * 110] = product;
  product = 0;
}